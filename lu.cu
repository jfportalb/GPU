
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <iostream>
using namespace std;

/**
 * O agumento deve ser double
 */
#define GET_TIME(now) { \
	struct timespec time; \
	clock_gettime(CLOCK_MONOTONIC_RAW, &time); \
	now = time.tv_sec + time.tv_nsec/1000000000.0; \
}

/**
 * Para checar erros em chamadas Cuda
 */
#define CUDA_SAFE_CALL(call) { \
	hipError_t err = call;     \
	if(err != hipSuccess) {    \
		fprintf(stderr,"Erro no arquivo '%s', linha %i: %s.\n",__FILE__, __LINE__,hipGetErrorString(err)); \
		exit(EXIT_FAILURE); } \
}

void luSeq (double *A, int n) {
	for (int i=0; i<n-1; i++){
		for (int j=1; j<n; j++){
			A[j*n+i] = A[j*n+i]/A[i*n+i];
			for (int k=1; k<n; k++){
				A[j*n+k] = A[j*n+k] - A[j*n+i]*A[i*n+k];
			}
		}
	}
}

__global__ void luCalcCol(double *A , int dim, int i) {
	__shared__  double  Aii;
	if (threadIdx.x == 0) {
		Aii = A[i*(dim +1)];
	}
	__syncthreads ();
	int j = blockIdx.x * blockDim.x + threadIdx.x + i + 1;
	if ( j < dim ) {
		A[ j*dim+i ] /= Aii;
	}
}

__global__ void luCalcSub(double *A, int dim , int i) {
	__shared__  double  a_ji[32];
	__shared__  double  a_ik[32];
	int j = blockDim.x * blockIdx.x + threadIdx.x + i + 1;
	int k = blockDim.y * blockIdx.y + threadIdx.y + i + 1;
	if (( threadIdx.y == 0) && (j < dim)) {
		a_ji[threadIdx.x] = A[ j*dim + i ];
	}
	if (( threadIdx.x == 0) && (k < dim)) {
		a_ik[threadIdx.y] = A[ i*dim + k ];
	}
	__syncthreads ();
	if ((j < dim) && (k < dim)) {
		A[ j*dim + k ] -= a_ji[threadIdx.x] * a_ik[threadIdx.y];
	}
}

void  luGPU(double *A, int n, int blockSize) {
	int i, n_blocos;
	for (i = 0; i < n-1; i++) {
		n_blocos = ((n-i-1) + blockSize -1) / blockSize;
		dim3  g_blocos(n_blocos, n_blocos);
		dim3  n_threads(blockSize,blockSize);
		luCalcCol <<< n_blocos, blockSize >>>(A, n, i);
		CUDA_SAFE_CALL(hipGetLastError());
		luCalcSub <<< g_blocos, n_threads >>>(A, n, i);
		CUDA_SAFE_CALL(hipGetLastError());
	}
}

void fillMatrix(double* A, int n){
	for (int i=0; i<n; i++){
		for (int j=0; j<n; j++){
			A[i*n+j] = (i+1)*(j+1);
		}
	}
}

void checkResults(double *mat1, double *mat2, int n){
	for (int i=0; i<n; i++) {
		for (int j=0; j<n; j++) {
			if (fabs(mat1[i*n+j] - mat2[i*n+j]) > 1e-5) {
				cerr << "Resultado incorreto em " << i << " x " << j << " -> " << mat1[i*n+j] << " " <<  mat2[i*n+j] << endl;
				exit(EXIT_FAILURE);
			}
		}
	}
}

void printResults(int n, double timeSeq, double timeCpuGpu, double timeRunPar, double timeGpuCpu){
	cout << n << ";" << timeSeq << ";" << timeCpuGpu << ";" << timeRunPar << ";" << timeGpuCpu << endl;
}

int  main(int argc, char** argv) {
	int n, blockSize;
	double *Aseq, *Apar, *Adevice;
	double begin, end, timeSeq, timeCpuGpu, timeRunPar, timeGpuCpu;
	
	if(argc < 3) {
		cerr << "Digite: "<< argv[0] <<" <Dimensão da matriz> <Dimensão do bloco>" << endl;
		exit(EXIT_FAILURE);
	}
	n = atol(argv[1]);
	blockSize = atol(argv[2]);
	
	size_t  quant_mem = n*n*sizeof(double);
	Aseq = (double *) malloc(quant_mem);
	if ( Aseq == NULL   ) {
		cerr << "Memoria  insuficiente" << endl;
		exit(EXIT_FAILURE);
	}
	Apar = (double *) malloc(quant_mem);
	if ( Apar == NULL   ) {
		cerr << "Memoria  insuficiente" << endl;
		exit(EXIT_FAILURE);
	}
	fillMatrix(Aseq, n);
	
	GET_TIME(begin);
	CUDA_SAFE_CALL(hipMalloc((void**) &Adevice, quant_mem));
	CUDA_SAFE_CALL(hipMemcpy(Aseq, Adevice, quant_mem, hipMemcpyDeviceToHost));
	GET_TIME(end);
	timeCpuGpu = end-begin;
	
	GET_TIME(begin);
	luGPU(Adevice, n, blockSize);
	GET_TIME(end);
	timeRunPar = end-begin;
	
	GET_TIME(begin);
	CUDA_SAFE_CALL(hipMemcpy(Apar, Adevice, quant_mem, hipMemcpyDeviceToHost));
	GET_TIME(end);
	timeGpuCpu = end-begin;
	
	GET_TIME(begin);
	luSeq(Aseq, n);
	GET_TIME(end);
	timeSeq = end-begin;
	
	CUDA_SAFE_CALL(hipFree(Adevice));
	free(Aseq);
	free(Apar);
	
	checkResults(Aseq, Apar, n);
	printResults(n, timeSeq, timeCpuGpu, timeRunPar, timeGpuCpu);
	
	CUDA_SAFE_CALL(hipDeviceReset());
	exit(EXIT_SUCCESS);
}
