#include <stdio.h>
#include <sys/time.h>
#include <iostream>
#include <fstream>
using namespace std;

/**
 * O argumento deve ser double
 */
#define GET_TIME(now) { \
	struct timespec time; \
	clock_gettime(CLOCK_MONOTONIC_RAW, &time); \
	now = time.tv_sec + time.tv_nsec/1000000000.0; \
}

/**
 * Para checar erros em chamadas Cuda
 */
#define CUDA_SAFE_CALL(call) { \
	hipError_t err = call;     \
	if(err != hipSuccess) {    \
		fprintf(stderr,"Erro no arquivo '%s', linha %i: %s.\n",__FILE__, __LINE__,hipGetErrorString(err)); \
		exit(EXIT_FAILURE); } \
}

void setupMatrix(double *A, int n){
	for (int i=0; i<n; i++){
		A[i] = 20;
		A[n*n-i-1]=30;
	}
	double t = 20, a = 10.0/n;
	for (int i=1; i<n-1; i++){
		t+=a;
		A[i*n] = t;
		for (int j=1; j<n1; j++){
			A[i*n+j]=20;
		}
		A[i*n + n - 1] = t;
	}
}

void print(double *A, int n){	
	for (int i=0; i<n; i++){
		for (int j=0; j<n; j++){
			cout << A[í*n+j] << '\t';
		}
		cout << endl;
	}
}

int  main(int argc, char** argv) {
	int n=0, blockSize;
	double *Aseq, *Apar, *Adevice;
	double begin, end, timeSeq, timeCpuGpu, timeRunPar, timeGpuCpu;
	char *inputFileName, *outputFileName;
	
	if(argc < 4) {
		cerr << "Digite: "<< argv[0] <<" <Dimensão do bloco> <Arquivo de entrada> <Arquivo de saída>" << endl;
		exit(EXIT_FAILURE);
	}
	blockSize = atol(argv[1]);
	inputFileName = argv[2];
	outputFileName = argv[3];
	
	ifstream infile (inputFileName, ios::binary);
	infile.read(reinterpret_cast<char *>(&n), sizeof(int));
	
	size_t  matBytes = n*n*sizeof(double);
	Aseq = (double *) malloc(matBytes);
	if ( Aseq == NULL   ) {
		cerr << "Memoria  insuficiente" << endl;
		exit(EXIT_FAILURE);
	}
	infile.read(reinterpret_cast<char *>(Aseq), matBytes);
	infile.close();
	
	Apar = (double *) malloc(matBytes);
	if ( Apar == NULL   ) {
		cerr << "Memoria  insuficiente" << endl;
		exit(EXIT_FAILURE);
	}
	
	GET_TIME(begin);
	CUDA_SAFE_CALL(hipMalloc((void**) &Adevice, matBytes));
	CUDA_SAFE_CALL(hipMemcpy(Aseq, Adevice, matBytes, hipMemcpyDeviceToHost));
	GET_TIME(end);
	timeCpuGpu = end-begin;
	
	GET_TIME(begin);
	luGPU(Adevice, n, blockSize);
	GET_TIME(end);
	timeRunPar = end-begin;
	
	GET_TIME(begin);
	CUDA_SAFE_CALL(hipMemcpy(Apar, Adevice, matBytes, hipMemcpyDeviceToHost));
	GET_TIME(end);
	timeGpuCpu = end-begin;
	
	GET_TIME(begin);
	luSeq(Aseq, n);
	GET_TIME(end);
	timeSeq = end-begin;
	
	CUDA_SAFE_CALL(hipFree(Adevice));
	free(Aseq);
	free(Apar);
	
	checkResults(Aseq, Apar, n);
	
	ofstream outfile (outputFileName, ios::binary);
	outfile.write(reinterpret_cast<char *>(&n), sizeof(int));
	outfile.write(reinterpret_cast<char *>(Aseq), matBytes);
	outfile.close();
	
	printResults(n, timeSeq, timeCpuGpu, timeRunPar, timeGpuCpu);
	
	CUDA_SAFE_CALL(hipDeviceReset());
	exit(EXIT_SUCCESS);
}
