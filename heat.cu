
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <iostream>
#include <fstream>
using namespace std;

#define ALPHA 19e-5
#define DELTA_T 120
#define ROUNDS 3*60*60/DELTA_T
#define DISTANCE 0.1

/**
 * O argumento deve ser double
 */
#define GET_TIME(now) { \
	struct timespec time; \
	clock_gettime(CLOCK_MONOTONIC_RAW, &time); \
	now = time.tv_sec + time.tv_nsec/1000000000.0; \
}

/**
 * Para checar erros em chamadas Cuda
 */
#define CUDA_SAFE_CALL(call) { \
	hipError_t err = call;     \
	if(err != hipSuccess) {    \
		fprintf(stderr,"Erro no arquivo '%s', linha %i: %s.\n",__FILE__, __LINE__,hipGetErrorString(err)); \
		exit(EXIT_FAILURE); } \
}

void setupMatrix(double *A, int n){
	for (int i=0; i<n; i++){
		A[i] = 20;
		A[n*n-i-1]=30;
	}
	double t = 20, a = 10.0/(n-1);
	for (int i=1; i<n-1; i++){
		t+=a;
		A[i*n] = t;
		for (int j=1; j<n-1; j++){
			A[i*n+j]=20;
		}
		A[i*n + n - 1] = t;
	}
}

__global__ void updateHeat(double *last, double *next , int n, int deltaT) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int pos = i*n + j;
	if (i==0 || i==n-1 || j==0 || j==n-1){
		next[pos] = last[pos];
	} else if (i < n && j < n){
		next[pos] = last[pos] + 
			(ALPHA*deltaT/(DISTANCE*DISTANCE))*(last[pos-1]+last[pos+1]+last[pos-n]+last[pos+n]-4*last[pos]);	
	}
}

void print(double *A, int n){	
	cout << "#############################################################" << endl;
	for (int i=0; i<n; i++){
		for (int j=0; j<n; j++){
			cout << A[i*n+j] << "  ";
		}
		cout << endl;
	}
	cout << "#############################################################" << endl;
}

void playRounds(double **AdevicePointer, int n, int blockSize, int rounds, int deltaT) {

	double *Atemp, *aux, *Adevice = AdevicePointer[0], *A;
	size_t matBytes = n*n*sizeof(double);
	A = (double *) malloc(matBytes);
	CUDA_SAFE_CALL(hipMalloc((void**) &Atemp, matBytes));
	
	int nBlocks = (n + blockSize -1) / blockSize;
	dim3  gBlocks(nBlocks, nBlocks);
	dim3 nThreads(blockSize,blockSize);
	
	for(int i=0; i<rounds; i++){
		updateHeat <<< gBlocks, nThreads >>>(Adevice, Atemp, n, deltaT);
		CUDA_SAFE_CALL(hipGetLastError());
		aux = Adevice;
		Adevice = Atemp;
		Atemp = aux;
		CUDA_SAFE_CALL(hipMemcpy(A, Adevice, matBytes, hipMemcpyDeviceToHost));
		print(A, n);
	}
	CUDA_SAFE_CALL(hipFree(Atemp));
	AdevicePointer[0] =Adevice;
}

void playRoundsSeq(double **APointer, int n, int rounds, int deltaT) {
		cout << "OwO" << endl;
	double *A = APointer[0], *Temp, *aux;
		cout << "OwO" << endl;
	size_t matBytes = n*n*sizeof(double);
		cout << "OwO" << endl;
	Temp = (double *) malloc(matBytes);
		cout << "OwO" << endl;
	for(int r=0; r<rounds; r++){
		cout << "OwO" << endl;
		for (int i = 0; i < n; ++i) {
		cout << "OwO" << endl;
			Temp[i*n] = A[i*n];
		cout << "OwO" << endl;
			for (int j = 1; j < n-1; ++j)	{
		cout << "OwO" << endl;
				int pos = i*n + j;
		cout << "OwO" << endl;
				if (i==0 || i==n-1){
					Temp[pos] = A[pos];
				} else if (i < n && j < n){
					Temp[pos] = A[pos] + 
						(ALPHA*deltaT/(DISTANCE*DISTANCE))*(A[pos-1]+A[pos+1]+A[pos-n]+A[pos+n]-4*A[pos]);	
				}
		cout << "OwO" << endl;
			}
		cout << "OwO" << endl;
			Temp[(i+1)*n-1] = A[(i+1)*n-1];
		}
		cout << "OwO" << endl;
		aux = A;
		A = Temp;
		Temp = aux;
		print(A, n);
	}
	free(Temp);
	APointer[0] = A;
}


void printResults(int n, double timeCpuGpu, double timeRunPar, double timeGpuCpu){
	cout << n << ";" << timeCpuGpu << ";" << timeRunPar << ";" << timeGpuCpu << endl;
}

int  main(int argc, char** argv) {
	int n=0, blockSize;
	double *A, *Adevice;
	double begin, end, timeCpuGpu = 0, timeRun = 0, timeGpuCpu = 0;	
	if(argc < 3) {
		cerr << "Digite: "<< argv[0] <<" <Dimensão da matriz> <Tempo total> <Delta T> [Dimensão do bloco]" << endl;
		exit(EXIT_FAILURE);
	}
	n = atol(argv[1]);
	int deltaT = atol(argv[3]);
	int rounds = atol(argv[2])/deltaT;

	size_t matBytes = n*n*sizeof(double);
	A = (double *) malloc(matBytes);
	if ( A == NULL   ) {
		cerr << "Memoria  insuficiente" << endl;
		exit(EXIT_FAILURE);
	}
	setupMatrix(A, n);
	print(A, n);
	if (argc > 3){
		blockSize = atol(argv[4]);
		GET_TIME(begin);
		CUDA_SAFE_CALL(hipMalloc((void**) &Adevice, matBytes));
		CUDA_SAFE_CALL(hipMemcpy(Adevice, A, matBytes, hipMemcpyHostToDevice));
		GET_TIME(end);
		timeCpuGpu = end-begin;
		
		GET_TIME(begin);
		playRounds(&Adevice, n, blockSize, rounds, deltaT);
		GET_TIME(end);
		timeRun = end-begin;
		
		GET_TIME(begin);
		CUDA_SAFE_CALL(hipMemcpy(A, Adevice, matBytes, hipMemcpyDeviceToHost));
		GET_TIME(end);
		timeGpuCpu = end-begin;
		// print(A, n);
		
		CUDA_SAFE_CALL(hipFree(Adevice));
	} else {
		cout << "OwO" << endl;
		GET_TIME(begin);
		cout << "OwO" << endl;
		playRoundsSeq(&A, n, rounds, deltaT);
		GET_TIME(end);
		timeRun = end-begin;
	}
	printResults(n, timeCpuGpu, timeRun, timeGpuCpu);
	free(A);
	CUDA_SAFE_CALL(hipDeviceReset());
	exit(EXIT_SUCCESS);
}
